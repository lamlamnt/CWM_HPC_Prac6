// In this assignment you will write a basic kernel where every thread 
// will write out to console string "Hello world!".
// You will also initialize GPU using cudaSetDevice() and also launch
// your "Hello world" kernel.

#include <stdio.h>
#include <stdlib.h>
// we have to include few more things
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

//----------------------------------------------------------------------
// TASK 2: Write a "Hello world" kernel
// 
// Remember that kernel is defined by __global__ and inside it looks like
// a serial code for CPU. For printing out to console you can use printf().

// write your kernel here
__global__ void helloworld_GPU(void){
printf("Hello world!\n");
}

//----------------------------------------------------------------------

int main(void) {
  //----------------------------------------------------------------------
  // TASK 1: Initiate GPU using cudaSetDevice()
  //
  // You can also try to write a check if there is a device with that id,
  // so the code behaves nicely when it fails
  
  // write you GPU initialization here
	
  //----------------------------------------------------------------------

	int deviceid = 0; // using GPU with id 0
	int devCount;
	// gets number of GPU available
	hipGetDeviceCount(&devCount);
	// check if we have enough GPUs
	if(deviceid<devCount) {
	// tell CUDA that we want to use GPU 0
		hipSetDevice(deviceid);
	}
	else return(1);
	// run CUDA kernel
	helloworld_GPU<<<1,1>>>();
  //----------------------------------------------------------------------
  // TASK 3: execute your "Hello world" kernel on 1 block with 5 threads 
  //         using execution configuration syntax.
  // 
  // You may use whatever syntax version you prefer, a simplified one 
  // dimensional or full three dimensional call using dim3 data type.
  
  // execute your "Hello world" kernel here
  
  //----------------------------------------------------------------------

  hipDeviceReset(); 
  return (0);
}
